#include "hip/hip_runtime.h"
/* CUDA blur
 * Kevin Yuh, 2014 */

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_convolve_cuda.cuh"


/* 
Atomic-max function. You may find it useful for normalization.

We haven't really talked about this yet, but __device__ functions not
only are run on the GPU, but are called from within a kernel.

Source: 
http://stackoverflow.com/questions/17399119/
cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
*/
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}



__global__
void
cudaProdScaleKernel(const hipfftComplex *raw_data, const hipfftComplex *impulse_v, 
    hipfftComplex *out_data,
    int padded_length) {


    /* TODO ok: Implement the point-wise multiplication and scaling for the
    FFT'd input and impulse response. 

    Recall that these are complex numbers, so you'll need to use the
    appropriate rule for multiplying them. 

    Also remember to scale by the padded length of the signal
    (see the notes for Question 1).

    As in Assignment 1 and Week 1, remember to make your implementation
    resilient to varying numbers of threads.

    */
    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    
    // if (thread_index == 1)
    //     printf("%d\n", &padded_length);
    

    while (thread_index < padded_length) {
        out_data[thread_index].x = raw_data[thread_index].x * impulse_v[thread_index].x - raw_data[thread_index].y * impulse_v[thread_index].y;
        out_data[thread_index].x = out_data[thread_index].x / padded_length;
        out_data[thread_index].y = raw_data[thread_index].x * impulse_v[thread_index].y + raw_data[thread_index].y * impulse_v[thread_index].x;
        out_data[thread_index].y = out_data[thread_index].y / padded_length;
        // if (thread_index == 1){
        //     printf("%f\n", &raw_data[thread_index].x);
        //     printf("%f\n", &impulse_v[thread_index].x);
        //     printf("%f\n", &out_data[thread_index].x);

        // }
        thread_index += blockDim.x * gridDim.x;
    }

}

__global__
void
cudaMaximumKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2 ok: Implement the maximum-finding and subsequent
    normalization (dividing by maximum).

    There are many ways to do this reduction, and some methods
    have much better performance than others. 

    For this section: Please explain your approach to the reduction,
    including why you chose the optimizations you did
    (especially as they relate to GPU hardware).

    You'll likely find the above atomicMax function helpful.
    (CUDA's atomicMax function doesn't work for floating-point values.)
    It's based on two principles:
        1) From Week 2, any atomic function can be implemented using
        atomic compare-and-swap.
        2) One can "represent" floating-point values as integers in
        a way that preserves comparison, if the sign of the two
        values is the same. (see http://stackoverflow.com/questions/
        29596797/can-the-return-value-of-float-as-int-be-used-to-
        compare-float-in-cuda)

    */
/*
    allocate shared memory for 1024 floats, because the max number of thread per 
    block is 1024 for this hardware. 

    set the blockDim.x = 1024


*/
    __shared__ float data[1024];
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    data[threadIdx.x] = 0;

    while (i < padded_length) {
            // if (i == 100) printf("%d max\n", &i);
        data[threadIdx.x] = out_data[i].x;
        __syncthreads();
        int l = blockDim.x;
        while (l > 1) {
            int bias = l / 2;
            while (threadIdx.x < bias) {
                data[threadIdx.x] = (fabs(data[threadIdx.x])>fabs(data[threadIdx.x + bias]))? \
                        data[threadIdx.x]:data[threadIdx.x + bias];
                __syncthreads();
            }
            l /= 2;
        }
        atomicMax(max_abs_val, fabs(data[0]));
        i += blockDim.x * gridDim.x;
    }
}

__global__
void
cudaDivideKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2 ok: Implement the division kernel. Divide all
    data by the value pointed to by max_abs_val. 

    This kernel should be quite short.
    */
    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    while (thread_index < padded_length) {
        // if (thread_index == 100) printf("%d divide\n", &thread_index);
        out_data[thread_index].x /= *max_abs_val;
        thread_index += blockDim.x * gridDim.x;
    }

}


void cudaCallProdScaleKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const hipfftComplex *raw_data,
        const hipfftComplex *impulse_v,
        hipfftComplex *out_data,
        const unsigned int padded_length) {
        
    /* TODO ok Call the element-wise product and scaling kernel. */
    cudaProdScaleKernel<<<blocks, threadsPerBlock>>>(raw_data, impulse_v, out_data, padded_length);
}

void cudaCallMaximumKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        

    /* TODO 2 ok: Call the max-finding kernel. */
    cudaMaximumKernel<<<blocks, threadsPerBlock>>>(out_data, max_abs_val, padded_length);
}


void cudaCallDivideKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        
    /* TODO 2 ok: Call the division kernel. */
    cudaDivideKernel<<<blocks, threadsPerBlock>>>(out_data, max_abs_val, padded_length);
}
